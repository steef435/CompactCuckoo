#include "hip/hip_runtime.h"
#include <atomic>

#include "bcht.hpp"
#include "HashTableGeneric.cuh"

#ifndef NUMGEN
#define NUMGEN
#include "numbergenerators.cu"
#endif

GPUHEADER_G
void readList(uint64_cu* xs, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    uint64_cu val = 0;

    for (int i = index; i < N; i+= stride) {
        val += xs[i];
    }
}

// TODO: this does not work, needs different readList impl.
//void warmupThreads(int T, uint64_cu* xs, int N, int numLoops) {
//    std::vector<std::thread> vecThread(T);
//    for (int i = 0; i < T; i++) {
//        vecThread.at(i) = std::thread(readList, xs, N, numLoops, T, i);
//    }
//
//    //Join Threads
//    for (int i = 0; i < T; i++) {
//        vecThread.at(i).join();
//    }
//}

void BenchmarkGeneralFilling(int NUM_TABLES_start, int NUM_TABLES, int INTERVAL, int NUM_SAMPLES, int NUM_THREADS, int NUM_LOOPS, int LOOP_STEP, int NUM_HASHES, int HASH_STEP, int NUM_REHASHES,
    int PERCENTAGE, int P_STEPSIZE, int DEPTH, std::vector<std::string>* params = nullptr) {

    const int WARMUP = 0;

    printf("=====================================================================\n");
    printf("                   Starting INSERTION  GENERAL BENCHMARK             \n");
    printf("=====================================================================\n");

    std::ofstream myfile;
    std::string specifier = "";
#ifdef GPUCODE
    specifier += "-GPU";
#else
    specifier += "-CPU";
#endif
#ifdef REHASH
    specifier += "-REHASH";
#endif
    std::string filename = "results/benchfill" + specifier + ".csv";

    if (params) {
        printf("Opening\n");
        myfile.open(filename, std::ios_base::app);
        printf("Maybe\n");
    }
    else {
        myfile.open(filename);
    }

    if (!myfile.is_open()) {
        printf("File %s Failed to Open\n", filename.c_str());

        return;
    }
    printf("File Opened\n");

    if (!params) {
        myfile << "tablesize,numthreads,loops,hashes,rehashes,collision_percentage,collision_depth,samples,type,interval,time,test\n";
    }

    printf("=====================================================================\n");
    printf("                     Starting Cleary-Cuckoo                \n\n");

    //Tablesizes
    bool setup = true;
    for (int N = NUM_TABLES_start; N < NUM_TABLES_start + NUM_TABLES; N++) {
        if (params && setup) {
            N = std::stoi(params->at(0));
        }
        printf("Table Size:%i\n", N);

        int size = std::pow(2, N);
        int setsize = (int)(size / INTERVAL);
        int lookupSize = size / 4;

        if (setsize == 0) {
            printf("Error: Number of Intervals is greater than number of elements\n");
        }

        //Number of Threads
        //TODO change start to 0
        for (int T = 1; T < NUM_THREADS; T++) {
            if (params && setup) {
                T = std::stoi(params->at(1));
            }
            printf("\tNumber of Threads:%i\n", T);
            //Number of Loops
            for (int L = 0; L < NUM_LOOPS; L+= LOOP_STEP) {
#ifdef GPUCODE
                int numThreads = std::pow(2, T);
#else
                int numThreads = T + 1;
#endif

                if (params && setup) {
                    L = std::stoi(params->at(2));
                }
                printf("\t\tNumber of Loops:%i\n", L);
                //Number of Hashes
                for (int H = 1; H <= NUM_HASHES; H+=HASH_STEP) {
                    printf("\t\t\tNumber of Hashes:%i\n", H);
                    if (params && setup) {
                        H = std::stoi(params->at(3));
                    }

                    for (int R = 0; R <= NUM_REHASHES; R++) {
                        printf("\t\t\t\tRehashes:%i\n", R);
                        for (int P = 0; P <= PERCENTAGE; P += P_STEPSIZE) {
                            printf("\t\t\t\t\tPercentage:%i\n", P);
                            for (int D = 1; D <= DEPTH; D += 1) {
                                printf("\t\t\t\t\t\tDepth:%i\n", D);
                                //Number of samples
                                for (int S = 0; S < NUM_SAMPLES; S++) {
                                    printf("\t\t\t\t\t\t\tSample:%i\n", S);
                                    if (params && setup) {
                                        S = std::stoi(params->at(4));
                                    }
                                    setup = false;
                                    //Init Cleary Cuckoo

    #ifdef GPUCODE
                                    //printf("InitTable\n");
                                    ClearyCuckoo* cc;
                                    //printf("\tAllocTable\n");
                                    gpuErrchk(hipMallocManaged((void**)&cc, sizeof(ClearyCuckoo)));
                                    //printf("\tStartTable\n");
                                    new (cc) ClearyCuckoo(N, H);
                                    //printf("InitFailFlag\n");
                                    int* failFlag;
                                    gpuErrchk(hipMallocManaged((void**)&failFlag, sizeof(int)));
                                    (*failFlag) = false;
    #else
                                    ClearyCuckoo* cc = new ClearyCuckoo(N, H);
                                    int* failFlag = new int;
                                    (*failFlag) = false;
    #endif
                                    //printf("SetVals\n");
                                    cc->setMaxLoops(L);
                                    cc->setMaxRehashes(R);
                                    //printf("getHashlistCopy\n");
                                    int* hs = cc->getHashlistCopy();
                                    //printf("Generate CollisionSet\n");
                                    uint64_cu* vals = generateCollisionSet(size, N, H, hs, P, D);
                                    delete[] hs;
                                    //printf("Numsgenned\n");

                                    //printf("vals:\n");


                                    //Warmup
                                    //printf("Warmup\n");
                                    //readList(vals, size, 20);
                                    // cc->readEverything(size * 50);
                                    // warmupThreads(numThreads, vals, size, 20);



                                    //printf("Starting\n");
                                    //Loop over intervals
                                    for (int j = 0; j < INTERVAL + WARMUP; j++) {
                                        //Fill the table
                                        std::chrono::steady_clock::time_point begin;
                                        std::chrono::steady_clock::time_point end;

                                        if (j < WARMUP) {
                                            //cc->readEverything(20);
                                        }

                                        if (j >= WARMUP && !(*failFlag)) {
                                            //printf("Insertion %i\n", j);
                                            begin = std::chrono::steady_clock::now();
                                            fillClearyCuckoo << <1, std::pow(2, T) >> > (setsize, vals, cc, failFlag, setsize * (j - WARMUP));
                                            gpuErrchk(hipPeekAtLastError());
                                            gpuErrchk(hipDeviceSynchronize());

                                            //End the timer
                                            end = std::chrono::steady_clock::now();

                                            myfile << N << "," << numThreads << "," << L << "," << H << "," << R << "," << P << "," << D << "," << S << ",cuc," << (j - WARMUP) << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) / setsize << ", INS, \n";
                                        }

                                        if (failFlag) {
                                            myfile << N << "," << numThreads << "," << L << "," << H << "," << R << "," << P << "," << D << "," << S << ",cuc," << (j - WARMUP) << "," << -1 << ", INS,\n";
                                        }

                                        //Lookup Time Test
                                        if (j >= WARMUP && !(*failFlag)) {
                                            begin = std::chrono::steady_clock::now();
                                            lookupClearyCuckoo << <1, std::pow(2, T) >> > (lookupSize, 0, setsize * (j - WARMUP + 1), vals, cc);
                                            gpuErrchk(hipPeekAtLastError());
                                            gpuErrchk(hipDeviceSynchronize());

                                            //End the timer
                                            end = std::chrono::steady_clock::now();

                                            myfile << N << "," << numThreads << "," << L << "," << H << "," << R << "," << P << "," << D << "," << S << ",cuc," << (j - WARMUP) << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) / lookupSize << ", LOOK, \n";
                                        }

                                    }
                                    gpuErrchk(hipFree(cc));
                                    gpuErrchk(hipFree(failFlag));
                                    gpuErrchk(hipFree(vals));
                                }
                            }
                        }
                    }
                }
            }
        }
    }

    myfile.close();
    printf("\nBenchmark Done\n");
}


void BenchmarkSpeed(int NUM_TABLES_start, int NUM_TABLES, int INTERVAL, int NUM_SAMPLES, int NUM_THREADS, int PERCENTAGE, int P_STEPSIZE, int DEPTH, bool clearyBool=true, std::string source="") {

    const int WARMUP = 0;

    printf("=====================================================================\n");
    printf("                     Starting INSERTION  BENCHMARK                    \n");
    printf("=====================================================================\n");

    std::ofstream myfile;
    std::string specifier = "";
#ifdef GPUCODE
    specifier += "-GPU";
#else
    specifier += "-CPU";
#endif
#ifdef REHASH
    specifier += "-REHASH";
#endif
    if (clearyBool) {
        specifier += "-CLEARY";
    }

    if (source != "") {
        specifier += "-REALDATA-";

        std::string filename;

        size_t pos = source.find_last_of("/");
        if(pos != std::string::npos)
          filename.assign(source.begin() + pos + 1, source.end());
        else
          filename = source;

        specifier += filename;

    }

    std::string filename = "results/benchspeed" + specifier + ".csv";

    myfile.open(filename);

    if (!myfile.is_open()) {
        printf("File %s Failed to Open\n", filename.c_str());

        return;
    }
    printf("File Opened\n");

    //Use file data
    uint64_cu* loadedvals = nullptr;
    uint64_cu* loadedvals32 = nullptr;
    int loadedsize = 0;
    if (source != "") {
        //printf("Reading Data\n");
        loadedvals = readCSV(source, &loadedsize);
        loadedvals32 = moduloList(loadedvals, loadedsize, std::pow(2,28));
    }

    myfile << "tablesize,numthreads,collision_percentage,collision_depth,samples,type,interval,time,test\n";

    printf("=====================================================================\n");
    //printf("                     Starting Cleary-Cuckoo                \n\n");

    //Tablesizes
    for (int N = NUM_TABLES_start; N < NUM_TABLES_start + NUM_TABLES; N++) {
        // Set GPU constants
        hipMemcpyToSymbol(HIP_SYMBOL(AS), &N, sizeof(int));

        int size = std::pow(2, N);
        int setsize = (int)(size / INTERVAL);
        int lookupSize = size / 4;

        printf("Table Size:%i (%i)\n", N, size);

        if (setsize == 0) {
            printf("Error: Number of Intervals is greater than number of elements\n");
        }

        if(loadedvals != nullptr && loadedsize < size){
          printf("\t Breaking: Table size larger than input size\n");
          break;
        }

        // Generate / read input data
        uint64_cu* vals;
        uint64_cu* vals32;
        if (loadedvals == nullptr) {
            //vals = generateCollisionSet(size, N, NHASHFUNCTIONS, hs, P, D);
            vals32 = generateRandomSet(size, std::pow(2, DATASIZENEW));
            vals = vals32;
        }
        else {
            vals = loadedvals;
            vals32 = loadedvals32;
        }

        //Number of Thread blocks
        //int minThreads = (int)std::log2(std::max(TILESIZE, TILESIZE_CBUC));

        int MAX_BLOCK_SIZE = 9;
        int numThreadsBlock = std::pow(2, MAX_BLOCK_SIZE);
        int maxBlocks = (size + numThreadsBlock - 1) / numThreadsBlock;
        for (int numBlocks = maxBlocks / 10; numBlocks <= maxBlocks; numBlocks += (maxBlocks / 10)) { //T < std::min(N, minThreads+ NUM_THREADS); T++) {
            //int numThreads = 1;
            //int numBlocks = 1;

            // if (T > MAX_BLOCK_SIZE) {
            //     numThreads = std::pow(2, MAX_BLOCK_SIZE);
            //     numBlocks = std::pow(2, T - MAX_BLOCK_SIZE);
            // }
            // else {
            //     numThreads = std::pow(2, T);
            // }
            //int T = N;
            //numBlocks = (size + numThreadsBlock - 1) / numThreadsBlock;
            int numThreads = numBlocks * numThreadsBlock;

            printf("\tNumber of Threads: %i (%i blocks, %i threads)\n", numThreads, numBlocks, numThreadsBlock);

            bool compact;
            int N_new;
            for (int P = 0; P <= PERCENTAGE; P += P_STEPSIZE) {
                printf("\t\tPercentage:%i\n", P);
                for (int D = 1; D <= DEPTH; D += 1) {
                    printf("\t\t\tDepth:%i\n", D);
                    //Number of samples
                    for (int S = 0; S < NUM_SAMPLES; S++) {
                        printf("\t\t\t\tSample:%i\n", S);

                        /***********************************************************************************************
                        *
                        * ClearyCuckoo Speed Test
                        *
                        ***********************************************************************************************/
                        printf("\t\t\t\t\tClearyCuckoo\n");
                        // Set GPU constants
                        hipMemcpyToSymbol(HIP_SYMBOL(AS), &N, sizeof(int));
                        compact = true;
                        hipMemcpyToSymbol(HIP_SYMBOL(is_compact), &compact, sizeof(bool));

                        HashTableGeneric<uint32_t,1>* cc;
                        gpuErrchk(hipMallocManaged((void**)&cc, sizeof(HashTableGeneric<uint32_t,1>)));
                        new (cc) HashTableGeneric<uint32_t,1>(size);
                        int* failFlag;
                        gpuErrchk(hipMallocManaged((void**)&failFlag, sizeof(int)));
                        (*failFlag) = false;

                        int* tableCount;
                        gpuErrchk(hipMallocManaged((void**)&tableCount, sizeof(int)));

                        (*tableCount) = 0;

                        int* hs = new int[3];
                        for (int hsi = 0; hsi < NHASHFUNCTIONS; hsi++) {
                            hs[hsi] = hsi;
                        }

                        //delete[] hs;

                        //printf("Numsgenned\n");

                        //printf("vals:\n");


                        //Warmup
                        //printf("Warmup\n");
                        readList << <512, 256 >> > (vals, size);
                        readEverything<<<512,256>>>(size, cc);
                        // warmupThreads(numThreadsBlock, vals, size, 20);


                        //printf("Reading\n");
                        //Loop over intervals
                        int intervalMAX = INTERVAL + WARMUP;
                        if (loadedvals != nullptr) {
                            intervalMAX = std::min(loadedsize/setsize, intervalMAX);
                            //printf("SetSize %i\n", setsize);
                        }
                        //printf("intervalMAX %i", intervalMAX);
                        for (int j = 0; j < intervalMAX; j++) {
                            printf("\t\t\t\t\t\tInterval %i\n", j);
                            //Fill the table
                            std::chrono::steady_clock::time_point begin;
                            std::chrono::steady_clock::time_point end;

                            if (j < WARMUP) {
                                //cc->readEverything(20);
                            }

                            //Measurement Interval
                            if (j >= WARMUP && !(*failFlag)) {
                                begin = std::chrono::steady_clock::now();

                                //Do insertion iteration
                                int insertionSize = setsize;
                                //insertionSize = std::min((int)(size / INTERVAL), insertionSize);
                                //int insertionSize = size;

                                for (int k = 0; k < setsize; k+= insertionSize) {
                                    //printf("\t\t\t\t\t\tStartpoint: %i\n", setsize * (j - WARMUP) + k);
                                    fill<uint32_t,1> << <numBlocks, numThreadsBlock >> > (insertionSize, vals, cc, failFlag, setsize * (j - WARMUP) + k, tableCount);
                                    gpuErrchk(hipPeekAtLastError());
                                    gpuErrchk(hipDeviceSynchronize());

                                    // dupCheckClearyCuckoo << <numBlocks, numThreadsBlock >> > (insertionSize, vals, cc, setsize * (j - WARMUP) + k);
                                    // gpuErrchk(hipPeekAtLastError());
                                    // gpuErrchk(hipDeviceSynchronize());
                                }

                                //End the timer
                                end = std::chrono::steady_clock::now();

                                myfile << N << "," << numThreads << "," << P << "," << D << "," << S << ",ccuc," << j << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) << ", INS,\n";
                            }
                            if (*failFlag) {
                                //printf("\t\t\t\t\t\tFailed\n");
                                myfile << N << "," << numThreads << "," << P << "," << D << "," << S << ",ccuc," << j << "," << -1 << ",INS,\n";
                                break;
                            }

                            // // Lookup Time Test
                            // if(j >= WARMUP && !(*failFlag)) {
                            //     begin = std::chrono::steady_clock::now();
                            //     lookupClearyCuckoo << <numBlocks, numThreadsBlock>> > (lookupSize, 0, setsize*(j- WARMUP + 1), vals, cc);
                            //     gpuErrchk(hipPeekAtLastError());
                            //     gpuErrchk(hipDeviceSynchronize());

                            //     //End the timer
                            //     end = std::chrono::steady_clock::now();

                            //     myfile << N << "," << numThreads << "," << P << "," << D << "," << S << ",ccuc," << (int) (INTERVAL*(*tableCount)/ size) << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) / lookupSize << ",LOOK,\n";
                            // }

                            // cc->print();

                        }
                        printf("Count ClearyCuckoo: %d\n", cc->count());

                        //printf("Delete CC Vars\n");
                        cc->deleteT();
                        gpuErrchk(hipFree(cc));
                        gpuErrchk(hipFree(failFlag));

                        if (clearyBool) {
                            /***********************************************************************************************
                            *
                            * Cleary Speed Test
                            *
                            ***********************************************************************************************/
                            printf("\t\t\t\t\tCleary\n");
                            Cleary* c;
                            gpuErrchk(hipMallocManaged((void**)&c, sizeof(Cleary)));
                            new (c) Cleary(N, numThreads);

                            (*tableCount) = 0;


                            //Loop over intervals
                            for (int j = 0; j < intervalMAX; j++) {
                                printf("\t\t\t\t\t\tInterval %i\n", j);
                                std::chrono::steady_clock::time_point begin;
                                std::chrono::steady_clock::time_point end;

                                //Fill the table
                                if (j >= WARMUP) {
                                    begin = std::chrono::steady_clock::now();

                                    fillCleary << <numBlocks, numThreadsBlock >> > (setsize, vals, c, setsize * (j - WARMUP), tableCount);
                                    gpuErrchk(hipPeekAtLastError());
                                    gpuErrchk(hipDeviceSynchronize());

                                    //End the timer
                                    end = std::chrono::steady_clock::now();
                                    myfile << N << "," << numThreads << "," << P << "," << D << "," << S << ",cle," << j << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) << ",INS,\n";
                                }

                                //Lookup
                                if (j >= WARMUP) {
                                    begin = std::chrono::steady_clock::now();
                                    lookupCleary << <numBlocks, numThreadsBlock >> > (lookupSize, 0, setsize * (j - WARMUP + 1), vals, c);
                                    gpuErrchk(hipPeekAtLastError());
                                    gpuErrchk(hipDeviceSynchronize());

                                    //End the timer
                                    end = std::chrono::steady_clock::now();

                                    myfile << N << "," << numThreads << "," << P << "," << D << "," << S << ",cle," << j << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) << ",LOOK,\n";
                                }

                            }

                            delete (c);
                            gpuErrchk(hipFree(c));
                        }

                        /***********************************************************************************************
                        *
                        * Cuckoo Speed Test
                        *
                        ***********************************************************************************************/
                        printf("\t\t\t\t\tCuckoo\n");
                        // Set GPU constants
                        hipMemcpyToSymbol(HIP_SYMBOL(AS), &N, sizeof(int));
                        compact = false;
                        hipMemcpyToSymbol(HIP_SYMBOL(is_compact), &compact, sizeof(bool));
                        HashTableGeneric<uint64_cu,1>* cuc;
                        gpuErrchk(hipMallocManaged((void**)&cuc, sizeof(HashTableGeneric<uint64_cu,1>)));
                        new (cuc) HashTableGeneric<uint64_cu,1>(size);
                        int* failFlag2;
                        gpuErrchk(hipMallocManaged((void**)&failFlag2, sizeof(int)));
                        (*failFlag2) = false;

                        (*tableCount) = 0;

                        //Warmup
                        //printf("Warmup\n");
                        //readList(vals, size, 20);
                        readList << <512, 256 >> > (vals, size);
                        readEverything << <512, 256 >> > (size, cuc);
                        // warmupThreads(numThreadsBlock, vals, size, 20);


                        //printf("Reading\n");
                        //Loop over intervals
                        for (int j = 0; j < intervalMAX; j++) {
                            //Fill the table
                            std::chrono::steady_clock::time_point begin;
                            std::chrono::steady_clock::time_point end;

                            if (j < WARMUP) {
                                // TODO: needs rework
                                //cc->readEverything(20);
                            }

                            if (j >= WARMUP && !(*failFlag2)) {
                                printf("\t\t\t\t\t\tInterval %i\n", j);
                                begin = std::chrono::steady_clock::now();


                                //Do insertion iteration
                                int insertionSize = setsize;
                                //insertionSize = std::min((int)(size / INTERVAL), insertionSize);
                                //int insertionSize = size;
                                printf("Cuckoo insertion size: %u\n", insertionSize);

                                for (int k = 0; k < setsize; k += insertionSize) {
                                    //printf("\t\t\t\t\t\tStartpoint: %i\n", setsize * (j - WARMUP) + k);
                                    fill<uint64_cu,1> << <numBlocks, numThreadsBlock >> > (insertionSize, vals, cuc, failFlag2, setsize* (j - WARMUP) + k, tableCount);
                                    gpuErrchk(hipPeekAtLastError());
                                    gpuErrchk(hipDeviceSynchronize());

                                    // dupCheckCuckoo << <numBlocks, numThreadsBlock >> > (insertionSize, vals, cuc, setsize * (j - WARMUP) + k);
                                    // gpuErrchk(hipPeekAtLastError());
                                    // gpuErrchk(hipDeviceSynchronize());
                                }

                                //End the timer
                                end = std::chrono::steady_clock::now();

                                myfile << N << "," << numThreads << "," << P << "," << D << "," << S << ",cuc," << j << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) << ", INS,\n";
                            }
                            if (*failFlag2) {
                                //cuc->print();
                                myfile << N << "," << numThreads << "," << P << "," << D << "," << S << ",cuc," << j << "," << -1 << ",INS,\n";
                                break;
                            }

                            //Lookup Time Test
                            // if (j >= WARMUP && !(*failFlag2)) {
                            //     begin = std::chrono::steady_clock::now();
                            //     lookupCuckoo << <numBlocks, numThreadsBlock >> > (lookupSize, 0, setsize * (j - WARMUP + 1), vals, cuc);
                            //     gpuErrchk(hipPeekAtLastError());
                            //     gpuErrchk(hipDeviceSynchronize());

                            //     //End the timer
                            //     end = std::chrono::steady_clock::now();

                            //     myfile << N << "," << numThreads * numBlocks << "," << P << "," << D << "," << S << ",cuc," << (int) (INTERVAL*(*tableCount)/ size) << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) / lookupSize << ",LOOK,\n";
                            // }

                        }
                        printf("Count Cuckoo: %d\n", cuc->count());
                        //printf("Delete CC Vars\n");
                        cuc->deleteT();
                        gpuErrchk(hipFree(cuc));
                        gpuErrchk(hipFree(failFlag2));

                        /***********************************************************************************************
                        *
                        * ClearyCuckooBucketed Speed Test
                        *
                        ***********************************************************************************************/
                        printf("\t\t\t\t\tClearyCuckooBucketed\n");
                        // Set GPU constants
                        N_new = N - (int)std::log2(COMPACT_BUCKETSIZE);
                        hipMemcpyToSymbol(HIP_SYMBOL(AS), &N_new, sizeof(int));
                        compact = true;
                        hipMemcpyToSymbol(HIP_SYMBOL(is_compact), &compact, sizeof(bool));
                        HashTableGeneric<uint32_t,COMPACT_BUCKETSIZE>* ccb;
                        gpuErrchk(hipMallocManaged((void**)&ccb, sizeof(HashTableGeneric<uint32_t,COMPACT_BUCKETSIZE>)));
                        new (ccb) HashTableGeneric<uint32_t,COMPACT_BUCKETSIZE>(size);
                        int* failFlag3;
                        gpuErrchk(hipMallocManaged((void**)&failFlag3, sizeof(int)));
                        (*failFlag3) = false;
                        //printf("\t\t\t\t\t\tInitDone\n");

                        (*tableCount) = 0;

                        //Warmup
                        //printf("Warmup\n");
                        //readList(vals, size, 20);
                        readList << <512, 256 >> > (vals, size);
                        readEverything << <512, 256 >> > (size, ccb);
                        //warmupThreads(numThreadsBlock, vals, size, 20);


                        //printf("Reading\n");
                        //Loop over intervals
                        //printf("\t\t\t\t\t\tStartingLoops\n");
                        for (int j = 0; j < intervalMAX; j++) {
                            //Fill the table
                            std::chrono::steady_clock::time_point begin;
                            std::chrono::steady_clock::time_point end;

                            if (j < WARMUP) {
                                //cc->readEverything(20);
                            }

                            if (j >= WARMUP && !(*failFlag3)) {
                                printf("\t\t\t\t\t\tInterval %i\n", j);
                                begin = std::chrono::steady_clock::now();

                                //Do insertion iteration
                                int insertionSize = setsize;
                                //insertionSize = std::min((int)(size / INTERVAL), insertionSize);
                                //printf("\t\t\t\t\t\tInsertionSize: %i\n", insertionSize);
                                //int insertionSize = size;

                                for (int k = 0; k < setsize; k += insertionSize) {
                                    //printf("\t\t\t\t\t\tStartpoint: %i\n", setsize * (j - WARMUP) + k);
                                    fill<uint32_t,COMPACT_BUCKETSIZE> << <numBlocks, numThreadsBlock >> > (insertionSize, vals32, ccb, failFlag3, setsize * (j - WARMUP) + k, tableCount);
                                    gpuErrchk(hipPeekAtLastError());
                                    gpuErrchk(hipDeviceSynchronize());

                                    // dupCheckClearyCuckooBucketed<TILESIZE> << <numBlocks, numThreadsBlock >> > (insertionSize, vals32, ccb, setsize * (j - WARMUP) + k);
                                    // gpuErrchk(hipPeekAtLastError());
                                    // gpuErrchk(hipDeviceSynchronize());
                                }

                                //End the timer
                                end = std::chrono::steady_clock::now();

                                myfile << N << "," << numThreads << "," << P << "," << D << "," << S << ",ccbuc," << j << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) << ", INS,\n";
                            }
                            if (*failFlag3) {
                                myfile << N << "," << numThreads << "," << P << "," << D << "," << S << ",ccbuc," << j << "," << -1 << ",INS,\n";
                                break;
                            }
                            //printf("\t\t\t\t\t\tLookup\n");
                            //Lookup Time Test
                            // if (j >= WARMUP && !(*failFlag3)) {
                            //     begin = std::chrono::steady_clock::now();
                            //     lookupClearyCuckooBucketed<TILESIZE> << <numBlocks, numThreadsBlock >> > (lookupSize, 0, setsize * (j - WARMUP + 1), vals32, ccb);
                            //     gpuErrchk(hipPeekAtLastError());
                            //     gpuErrchk(hipDeviceSynchronize());

                            //     //End the timer
                            //     end = std::chrono::steady_clock::now();

                            //     myfile << N << "," << numThreads * numBlocks << "," << P << "," << D << "," << S << ",ccbuc," << (int) (INTERVAL*(*tableCount)/ size) << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) / lookupSize << ",LOOK,\n";
                            // }

                        }
                        printf("Count ClearyCuckoo bucketed: %d\n", ccb->count());
                        ccb->deleteT();
                        gpuErrchk(hipFree(ccb));
                        gpuErrchk(hipFree(failFlag3));

                        /***********************************************************************************************
                        *
                        * CuckooBucketed Speed Test
                        *
                        ***********************************************************************************************/
                        printf("\t\t\t\t\tCuckooBucketed\n");
                        // Set GPU constants
                        N_new = N - (int)std::log2(BUCKETSIZE);
                        hipMemcpyToSymbol(HIP_SYMBOL(AS), &N_new, sizeof(int));
                        compact = false;
                        hipMemcpyToSymbol(HIP_SYMBOL(is_compact), &compact, sizeof(bool));
                        HashTableGeneric<uint64_cu,BUCKETSIZE>* b;
                        gpuErrchk(hipMallocManaged((void**)&b, sizeof(HashTableGeneric<uint64_cu,BUCKETSIZE>)));
                        new (b) HashTableGeneric<uint64_cu,BUCKETSIZE>(size);
                        int* failFlag4;
                        gpuErrchk(hipMallocManaged((void**)&failFlag4, sizeof(int)));
                        (*failFlag4) = false;
                        //printf("\t\t\t\t\t\tInitDone\n");

                        (*tableCount) = 0;

                        //Warmup
                        //printf("Warmup\n");
                        //readList(vals, size, 20);
                        readList << <512, 256 >> > (vals, size);
                        readEverything << <512, 256 >> > (size, b);
                        //warmupThreads(numThreadsBlock, vals, size, 20);


                        //printf("Reading\n");
                        //Loop over intervals
                        //printf("\t\t\t\t\t\tStartingLoops\n");
                        for (int j = 0; j < intervalMAX; j++) {
                            printf("\t\t\t\t\t\tInterval %i\n", j);
                            //Fill the table
                            std::chrono::steady_clock::time_point begin;
                            std::chrono::steady_clock::time_point end;

                            if (j < WARMUP) {
                                //cc->readEverything(20);
                            }

                            if (j >= WARMUP && !(*failFlag4)) {
                                begin = std::chrono::steady_clock::now();

                                //Do insertion iteration
                                int insertionSize = setsize;
                                //insertionSize = std::min((int)(size / INTERVAL), insertionSize);
                                //printf("\t\t\t\t\t\tInsertionSize: %i\n", insertionSize);
                                //int insertionSize = size;

                                for (int k = 0; k < setsize; k += insertionSize) {
                                    //printf("\t\t\t\t\t\tStartpoint: %i\n", setsize * (j - WARMUP) + k);
                                    fill<uint64_cu,BUCKETSIZE> << <numBlocks, numThreadsBlock >> > (insertionSize, vals32, b, failFlag4, setsize * (j - WARMUP) + k, tableCount);
                                    gpuErrchk(hipPeekAtLastError());
                                    gpuErrchk(hipDeviceSynchronize());

                                    // dupCheckCuckooBucketed<TILESIZE_CBUC> << <numBlocks, numThreadsBlock >> > (insertionSize, vals32, b, setsize * (j - WARMUP) + k);
                                    // gpuErrchk(hipPeekAtLastError());
                                    // gpuErrchk(hipDeviceSynchronize());
                                }

                                //End the timer
                                end = std::chrono::steady_clock::now();

                                myfile << N << "," << numThreads << "," << P << "," << D << "," << S << ",buc," << j << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) << ", INS,\n";
                            }
                            if (*failFlag4) {
                                myfile << N << "," << numThreads << "," << P << "," << D << "," << S << ",buc," << j << "," << -1 << ",INS,\n";
                                break;
                            }
                            //printf("\t\t\t\t\t\tLookup\n");
                            //Lookup Time Test
                            // if (j >= WARMUP && !(*failFlag4)) {
                            //     begin = std::chrono::steady_clock::now();
                            //     lookupCuckooBucketed<TILESIZE_CBUC> << <numBlocks, numThreadsBlock >> > (lookupSize, 0, setsize * (j - WARMUP + 1), vals32, b);
                            //     gpuErrchk(hipPeekAtLastError());
                            //     gpuErrchk(hipDeviceSynchronize());

                            //     //End the timer
                            //     end = std::chrono::steady_clock::now();

                            //     myfile << N << "," << numThreads * numBlocks << "," << P << "," << D << "," << S << ",buc," << (int) (INTERVAL*(*tableCount)/ size) << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) / lookupSize << ",LOOK,\n";
                            // }

                        }

                        //ccb->print();

                        //printf("Delete CC Vars\n");
                        printf("Count Cuckoo bucketed: %d\n", b->count());
                        b->deleteT();
                        gpuErrchk(hipFree(b));
                        gpuErrchk(hipFree(failFlag4));
                        gpuErrchk(hipFree(tableCount));

                        /***********************************************************************************************
                        *
                        * bcht
                        *
                        ***********************************************************************************************/
                        // printf("\t\t\t\t\tBCHT\n");
                        // bght::bcht<int, int> bchtTable(((int) pow(2, N)), 0, 0);


                        // //Free any randomly generated datasets
                        // if (loadedvals == nullptr) {
                        //     gpuErrchk(hipFree(vals));
                        //     gpuErrchk(hipFree(vals32));
                        // }

                        // //Warmup
                        // //printf("Warmup\n");
                        // readList(vals, size, 20);
                        // //ccb->readEverything(size * 50);
                        // warmupThreads(numThreads, vals, size, 20);

                        // for (int j = 0; j < intervalMAX; j++) {
                        //     //Fill the table
                        //     std::chrono::steady_clock::time_point begin;
                        //     std::chrono::steady_clock::time_point end;

                        //     if (j < WARMUP) {
                        //         //cc->readEverything(20);
                        //     }

                        //     if (j >= WARMUP && !(*failFlag4)) {
                        //         begin = std::chrono::steady_clock::now();

                        //         //Do insertion iteration
                        //         int insertionSize = numThreads;
                        //         insertionSize = std::min((int)(size / INTERVAL), insertionSize);
                        //         //printf("\t\t\t\t\t\tInsertionSize: %i\n", insertionSize);

                        //         for (int k = 0; k < setsize; k += insertionSize) {
                        //             //printf("\t\t\t\t\t\tStartpoint: %i\n", setsize * (j - WARMUP) + k);
                        //             fillBCHT << <numBlocks, numThreads >> > (insertionSize, vals32, bchtTable, failFlag4, setsize * (j - WARMUP) + k, tableCount);
                        //             gpuErrchk(hipPeekAtLastError());
                        //             gpuErrchk(hipDeviceSynchronize());
                        //         }

                        //         //End the timer
                        //         end = std::chrono::steady_clock::now();

                        //         myfile << N << "," << numThreads * numBlocks << "," << P << "," << D << "," << S << ",bcht," << (int)(INTERVAL * (*tableCount) / size) << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) / setsize << ", INS,\n";
                        //     }
                        //     if (*failFlag4) {
                        //         myfile << N << "," << numThreads * numBlocks << "," << P << "," << D << "," << S << ",bcht," << (int)(INTERVAL * (*tableCount) / size) << "," << -1 << ",INS,\n";
                        //         break;
                        //     }
                        //     //printf("\t\t\t\t\t\tLookup\n");
                        //     //Lookup Time Test
                        //     if (j >= WARMUP && !(*failFlag4)) {
                        //         begin = std::chrono::steady_clock::now();
                        //         lookupBCHT << <numBlocks, numThreads >> > (lookupSize, 0, setsize * (j - WARMUP + 1), vals32, bchtTable);
                        //         gpuErrchk(hipPeekAtLastError());
                        //         gpuErrchk(hipDeviceSynchronize());

                        //         //End the timer
                        //         end = std::chrono::steady_clock::now();

                        //         myfile << N << "," << numThreads * numBlocks << "," << P << "," << D << "," << S << ",bcht," << (int)(INTERVAL * (*tableCount) / size) << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) / lookupSize << ",LOOK,\n";
                        //     }

                        // }
                    }
                }
            }
        }
        if (loadedvals == nullptr) {
          gpuErrchk(hipFree(vals32));
        }
    }

    //Free the loaded data set if exists
    if (loadedvals != nullptr) {
        gpuErrchk(hipFree(loadedvals));
        gpuErrchk(hipFree(loadedvals32));
    }

    myfile.close();
    printf("\nBenchmark Done\n");
}



void BenchmarkMaxOccupancy(int TABLE_START, int NUM_TABLES, int HASH_START, int NUM_HASHES, int HASH_STEP, int NUM_LOOPS, int LOOP_STEP, int NUM_REHASHES, int REHASH_STEP, int NUM_SAMPLES) {

    printf("=====================================================================\n");
    printf("                   Starting MAX Occupancy Benchmark                  \n");
    printf("=====================================================================\n");

    std::ofstream myfile;
    std::string specifier = "";
#ifdef GPUCODE
    specifier += "-GPU";
#else
    specifier += "-CPU";
#endif
#ifdef REHASH
    specifier += "-REHASH";
#endif

    std::string filename = "results/benchmax";
    if (HASH_START == 4 && NUM_HASHES == 1) {
        filename = "results/benchmax_4";
    }

    filename = filename + specifier + ".csv";


    myfile.open(filename);
    if (!myfile.is_open()) {
        printf("File %s Failed to Open\n", filename.c_str());
        return;
    }
    printf("File Opened");

    myfile << "tablesize,hashes,loops,rehashes,samples,max\n";

    //MAX_LOOPS
    for (int N = TABLE_START; N < TABLE_START + NUM_TABLES; N++) {
        printf("Table Size:%i\n", N);
        int size = std::pow(2, N);
        for (int H = HASH_START; H < HASH_START + NUM_HASHES; H+=HASH_STEP) {
            printf("\tNum of Hashes:%i\n", H);
            for (int L = 0; L < NUM_LOOPS; L+=LOOP_STEP) {
                printf("\t\tNum of Loops:%i\n", L);
                for (int R = 0; R <= NUM_REHASHES; R+=REHASH_STEP) {
                    printf("\t\t\tNum of Rehashes:%i\n", R);
                    for (int S = 0; S < NUM_SAMPLES; S++) {
                        //printf("\t\t'tSample Number:%i\n", S);
                        uint64_cu* vals = generateRandomSet(size);

                        //Init Cleary Cuckoo
                        //printf("INit Table\n");
#ifdef GPUCODE
                        ClearyCuckoo* cc;
                        gpuErrchk(hipMallocManaged((void**)&cc, sizeof(ClearyCuckoo)));
                        new (cc) ClearyCuckoo(N, H);
#else
                        ClearyCuckoo* cc = new ClearyCuckoo(N, H);
#endif
                        cc->setMaxLoops(L);
                        cc->setMaxRehashes(R);

                        //printf("INit Complete\n");
#ifdef GPUCODE
                        int* failFlag;
                        gpuErrchk(hipMallocManaged(&failFlag, sizeof(int)));
                        failFlag[0] = false;

                        //Var to store num of inserted values
                        addtype* occ;
                        gpuErrchk(hipMallocManaged(&occ, sizeof(addtype)));
                        occ[0] = 0;

                        fillClearyCuckoo << <1, 1 >> > (size, vals, cc, occ, failFlag);
                        gpuErrchk(hipPeekAtLastError());
                        gpuErrchk(hipDeviceSynchronize());

                        myfile << N << "," << H << "," << L << "," << R << "," << S << "," << occ[0] << ",\n";
#else
                        std::atomic<bool> failFlag(false);
                        std::atomic<addtype> occ(0);
                        //printf("Filling Table\n");
                        SpinBarrier barrier(1);

                        fillClearyCuckoo(size, vals, cc, &barrier, &occ, &failFlag);
                        //printf("Writing\n");
                        myfile << N << "," << H << "," << L << "," << R << "," << S << "," << occ.load() << ",\n";
#endif


#ifdef GPUCODE

                        gpuErrchk(hipFree(failFlag));
                        gpuErrchk(hipFree(cc));
                        gpuErrchk(hipFree(occ));
                        gpuErrchk(hipFree(vals));
#else
                        //printf("Deleting\n");
                        delete cc;
                        delete[] vals;
#endif
                        //printf("Done\n");
                    }
                }
            }
        }
    }

    myfile.close();

    printf("\n\nBenchmark Done\n");
}


void BenchmarkMaxOccupancyBucket(int TABLE_START, int NUM_TABLES, int HASH_START, int NUM_HASHES, int HASH_STEP, int NUM_LOOPS, int LOOP_STEP, int NUM_SAMPLES) {

    printf("=====================================================================\n");
    printf("                   Starting MAX Occupancy Benchmark Bucket               \n");
    printf("=====================================================================\n");

    std::ofstream myfile;
    std::string specifier = "";
#ifdef GPUCODE
    specifier += "-GPU";
#else
    specifier += "-CPU";
#endif
#ifdef REHASH
    specifier += "-REHASH";
#endif

    std::string filename = "results/benchmaxbucket";

    filename = filename + specifier + ".csv";


    myfile.open(filename);
    if (!myfile.is_open()) {
        printf("File %s Failed to Open\n", filename.c_str());
        return;
    }
    printf("File Opened\n");

    myfile << "tablesize,hashes,loops,samples,max\n";

    //MAX_LOOPS
    for (int N = TABLE_START; N < TABLE_START + NUM_TABLES; N++) {
        printf("Table Size:%i\n", N);
        int size = std::pow(2, N);
        for (int H = HASH_START; H < HASH_START + NUM_HASHES; H += HASH_STEP) {
            printf("\tNum of Hashes:%i\n", H);
            for (int L = 0; L < NUM_LOOPS; L += LOOP_STEP) {
                printf("\t\tNum of Loops:%i\n", L);

                for (int S = 0; S < NUM_SAMPLES; S++) {
                    //printf("\t\t'tSample Number:%i\n", S);
                    uint64_cu* vals = generateRandomSet(size, std::pow(2, DATASIZE_BUCKET));

                    //Init Cleary Cuckoo
                    //printf("INit Table\n");
#ifdef GPUCODE
                    ClearyCuckooBucketed<TILESIZE>* cc;
                    gpuErrchk(hipMallocManaged((void**)&cc, sizeof(ClearyCuckooBucketed<TILESIZE>)));
                    new (cc) ClearyCuckooBucketed<TILESIZE>(N, H);
#else
                    ClearyCuckooBucketed* cc = new ClearyCuckooBucketed<TILESIZE>(N, H);
#endif
                    cc->setMaxLoops(L);

                    //printf("INit Complete\n");
#ifdef GPUCODE
                    int* failFlag;
                    gpuErrchk(hipMallocManaged(&failFlag, sizeof(int)));
                    failFlag[0] = false;

                    //Var to store num of inserted values
                    addtype* occ;
                    gpuErrchk(hipMallocManaged(&occ, sizeof(addtype)));
                    occ[0] = 0;

                    fillClearyCuckooBucketed<TILESIZE> << <1, 16 >> > (size, vals, cc, occ, failFlag);
                    gpuErrchk(hipPeekAtLastError());
                    gpuErrchk(hipDeviceSynchronize());

                    myfile << N << "," << H << "," << L << "," << S << "," << occ[0] << ",\n";
#else
                    std::atomic<bool> failFlag(false);
                    std::atomic<addtype> occ(0);
                    //printf("Filling Table\n");
                    SpinBarrier barrier(1);

                    fillClearyCuckooBucketed<TILESIZE>(size * BS, vals, cc, &barrier, &occ, &failFlag);
                    //printf("Writing\n");
                    myfile << N << "," << H << "," << L << "," << S << "," << occ.load() << ",\n";
#endif


#ifdef GPUCODE

                    gpuErrchk(hipFree(failFlag));
                    gpuErrchk(hipFree(cc));
                    gpuErrchk(hipFree(occ));
                    gpuErrchk(hipFree(vals));
#else
                    //printf("Deleting\n");
                    delete cc;
                    delete[] vals;
#endif
                    //printf("Done\n");
                }
            }
        }
    }

    myfile.close();

    printf("\n\nBenchmark Done\n");
}


void BenchmarkBucketed(int NUM_BUCKETS, int BUCKET_SIZE, int INTERVAL, int NUM_SAMPLES, int NUM_THREADS, int NUM_LOOPS, int LOOP_STEP, int NUM_HASHES, int HASH_STEP, int NUM_REHASHES) {

    const int WARMUP = 0;

    printf("=====================================================================\n");
    printf("                   Starting INSERTION  GENERAL BENCHMARK             \n");
    printf("=====================================================================\n");

    std::ofstream myfile;
    std::string specifier = "";
#ifdef GPUCODE
    specifier += "-GPU";
#else
    specifier += "-CPU";
#endif
#ifdef REHASH
    specifier += "-REHASH";
#endif
    std::string filename = "results/BucketBenchfill" + specifier + ".csv";


    myfile.open(filename);

    if (!myfile.is_open()) {
        printf("File %s Failed to Open\n", filename.c_str());

        return;
    }
    printf("File Opened\n");

    myfile << "tablesize,numthreads,loops,hashes,buckets,bucketsize,samples,type,interval,time,test\n";


    printf("=====================================================================\n");
    printf("                     Starting Cleary-Cuckoo                \n\n");

    //Tablesizes
    for (int B = 1; B < NUM_BUCKETS; B++) {
        printf("NUM BUCKETS: %i\n", B);
        for (int Bs = 1; Bs < BUCKET_SIZE; Bs++) {
            printf("\tBUCKETSIZE: %i\n", Bs);
            int size = B * Bs;
            int lookupSize = size / 4;
            int setsize = (int)(size / INTERVAL);
            for (int H = 1; H < NUM_HASHES; H += HASH_STEP) {
                printf("\t\tNUM HASHES: %i\n", H);
                for (int L = 1; L < NUM_LOOPS; L += LOOP_STEP) {
                    printf("\t\t\tNUM LOOPS: %i\n", L);
                    //Number of Threads
                    for (int T = 0; T < NUM_THREADS; T++) {
                        printf("\t\t\t\tNumber of Threads:%i\n", T);
#ifdef GPUCODE
                        int numThreads = std::pow(2, T);
#else
                        int numThreads = T + 1;
#endif
                        for (int S = 0; S < NUM_SAMPLES; S++) {
                            uint64_cu* vals = generateRandomSet(size);

                            printf("\t\t\t\tSample:%i\n", S);
                            //Init Cleary Cuckoo Bucketed
#ifdef GPUCODE
                            ClearyCuckooBucketed<TILESIZE>* cc;
                            gpuErrchk(hipMallocManaged((void**)&cc, sizeof(ClearyCuckooBucketed<TILESIZE>)));
                            new (cc) ClearyCuckooBucketed<TILESIZE>(B, H);

                            int* failFlag;
                            gpuErrchk(hipMallocManaged((void**)&failFlag, sizeof(int)));
                            (*failFlag) = false;
#else
                            ClearyCuckooBucketed* cc = new ClearyCuckooBucketed<TILESIZE>(B, Bs, H);

                            int* failFlag = new int;
                            (*failFlag) = false;
#endif
                            cc->setMaxLoops(L);

                            for (int j = 0; j < INTERVAL + WARMUP; j++) {
                                //Fill the table
                                std::chrono::steady_clock::time_point begin;
                                std::chrono::steady_clock::time_point end;

                                if (j < WARMUP) {
                                    //cc->readEverything(20);
                                }

                                if (j >= WARMUP && !(*failFlag)) {
                                    //printf("Insertion %i\n", j);
                                    begin = std::chrono::steady_clock::now();
#ifdef GPUCODE
                                    fillClearyCuckooBucketed<TILESIZE> << <1, std::pow(2, T) >> > (setsize, vals, cc, failFlag, setsize * (j - WARMUP));
                                    gpuErrchk(hipPeekAtLastError());
                                    gpuErrchk(hipDeviceSynchronize());
#else
                                    std::vector<std::thread> vecThread(numThreads);
                                    SpinBarrier barrier(numThreads);
                                    //printf("LAUNCHING THREADS\n");
                                    for (int i = 0; i < numThreads; i++) {
                                        //printf("Starting Threads\n");
                                        vecThread.at(i) = std::thread(static_cast<void(*)(int, uint64_cu*, ClearyCuckoo*, SpinBarrier*, int*, addtype, int, int)>(fillClearyCuckooBucketed<TILESIZE>), setsize, vals, cc, &barrier, failFlag, setsize * (j - WARMUP), i, numThreads);
                                    }

                                    //Join Threads
                                    for (int i = 0; i < numThreads; i++) {
                                        vecThread.at(i).join();
                                    }
#endif
                                    //End the timer
                                    end = std::chrono::steady_clock::now();

                                    myfile << size << "," << numThreads << "," << L << "," << H << "," << B << "," << Bs << ","  << S << ",buc," << (j - WARMUP) << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) / setsize << ", INS, \n";
                                }

                                if (failFlag) {
                                    myfile << size << "," << numThreads << "," << L << "," << H << "," << B << "," << Bs << "," << S << ",buc," << (j - WARMUP) << "," << -1 << ", INS,\n";
                                }

                                //Lookup Time Test
                                if (j >= WARMUP && !(*failFlag)) {
                                    begin = std::chrono::steady_clock::now();
#ifdef GPUCODE
                                    lookupClearyCuckooBucketed<TILESIZE> << <1, std::pow(2, T) >> > (lookupSize, 0, setsize * (j - WARMUP + 1), vals, cc);
                                    gpuErrchk(hipPeekAtLastError());
                                    gpuErrchk(hipDeviceSynchronize());
#else
                                    std::vector<std::thread> vecThread(numThreads);
                                    for (int i = 0; i < numThreads; i++) {
                                        //printf("Starting Threads\n");
                                        vecThread.at(i) = std::thread(lookupClearyCuckooBucketed<TILESIZE>, lookupSize, 0, setsize * (j - WARMUP + 1), vals, cc, i, numThreads);
                                    }

                                    //Join Threads
                                    for (int i = 0; i < numThreads; i++) {
                                        vecThread.at(i).join();
                                    }
#endif
                                    //End the timer
                                    end = std::chrono::steady_clock::now();

                                    myfile << size << "," << numThreads << "," << L << "," << H << "," << B << "," << Bs << "," << S << ",cuc," << (j - WARMUP) << "," << (std::chrono::duration_cast<std::chrono::nanoseconds> (end - begin).count()) / lookupSize << ", LOOK, \n";
                                }

                            }


#ifdef GPUCODE

                            gpuErrchk(hipFree(failFlag));
                            gpuErrchk(hipFree(cc));
                            gpuErrchk(hipFree(vals));
#else
                            //printf("Deleting\n");
                            delete cc;
                            delete[] vals;
#endif
                            //printf("Done\n");
                        }
                    }
                }
            }
        }
    }
        myfile.close();
        printf("\nBenchmark Done\n");
}
